#include "awnn/tensor.h"
#include "awnndevice/device_utils.cuh"

#ifdef GLOBAL_COUNT_TENSOR_ALLOC_DEALLOC
#include "awnn/memory.h"
#endif


#include <hip/hip_runtime_api.h>  // hipMemset

void* mem_alloc_device(size_t size) {
  T* d_data;
  hipError_t cudaStat;
  cudaStat = hipMalloc(&d_data, size);
  AWNN_CHECK_EQ(cudaStat, hipSuccess);
#ifdef GLOBAL_COUNT_TENSOR_ALLOC_DEALLOC
  INC_TOTAL_TENSOR_ALLOC_DEVICE();
#endif
  return d_data;
}

void mem_free_device(void* d_data) {
  if (d_data) {
    hipFree(d_data);
#ifdef GLOBAL_COUNT_TENSOR_ALLOC_DEALLOC
    INC_TOTAL_TENSOR_DEALLOC_DEVICE();
#endif
  }
}

tensor_t tensor_make_device(uint const shape[], uint const ndims) {
  tensor_t t_device;

  uint capacity = 1;
  uint i;
  for (i = 0; i < ndims; ++i) {
    capacity *= shape[i];
    t_device.dim.dims[i] = shape[i];
  }
  for (; i < MAX_DIM; ++i) {
    t_device.dim.dims[i] = 0;
  }

  T* d_data = (T*)mem_alloc_device(capacity * sizeof(T));  // raw data at gpu mem in flat format

  t_device.data = d_data;
  t_device.mem_type = GPU_MEM;

  return t_device;
}

tensor_t tensor_make_zeros_device(uint const shape[], uint const ndims) {
  tensor_t t = tensor_make_device(shape, ndims);
  hipMemset(t.data, 0, tensor_get_capacity(t) * sizeof(T));

  return t;
}

tensor_t tensor_make_copy_h2d(tensor_t t_host) {
  uint capacity = tensor_get_capacity(t_host);
  T* d_data = (T*)mem_alloc_device(
      capacity * sizeof(T));  // raw data at gpu mem in flat format
  hipMemcpy(d_data, t_host.data, capacity * sizeof(T), hipMemcpyHostToDevice);

  tensor_t t_device;
  t_device.data = d_data;
  t_device.dim = t_host.dim;
  t_device.mem_type = GPU_MEM;

  return t_device;
}

void tensor_copy_d2h(tensor_t t_host, tensor_t t_device) {
  assert(t_device.mem_type == GPU_MEM);
  assert(t_host.mem_type == CPU_MEM);

  uint capacity = tensor_get_capacity(t_device);
  AWNN_CHECK_EQ(tensor_get_capacity(t_host), capacity)
  hipMemcpy(t_host.data, t_device.data, capacity * sizeof(T),
             hipMemcpyDeviceToHost);
}

void tensor_destroy_device(tensor_t* ptr_t_device) {
  assert(ptr_t_device->mem_type == GPU_MEM);

  mem_free_device(ptr_t_device->data);
}
