#include "hip/hip_runtime.h"
#include "awnn/common.h"

#include "awnndevice/cublas_wrappers.cuh"
#include "awnndevice/device_utils.cuh"
#include "awnndevice/layer_sandwich_device.cuh"
#include "awnndevice/layer_conv_device.cuh"

void layer_context_destroy_device(struct layer_context_device *context){
  tensor_destroy_device(&context->d_tmp);
  tensor_destroy_device(&context->d_dtmp);
}

__global__ void do_device_relu_forward(tensor_t d_x, tensor_t d_y){
  for (uint i : grid_stride_range(0u, d_capacity(d_x))) {
    d_y.data[i] = d_x.data[i] > 0 ? d_x.data[i] : 0.0;
  }
}
__global__ void do_device_relu_backward(tensor_t d_dx, tensor_t d_x, tensor_t d_dy){

  for (uint i : grid_stride_range(0u, d_capacity(d_x))) {
    d_dx.data[i] = d_x.data[i] > 0 ? d_dy.data[i] : 0.0;
  }
}

status_t relu_forward_device(tensor_t const d_x,
                                  lcache_t* cache,
                                  tensor_t d_y) {
  do_device_relu_forward<<<32, 1024>>>(d_x, d_y);

  if(cache){
    lcache_push(cache, d_x);
  }

  // print_tensor_device<<<1,1>>>(d_x);

  return S_OK;
}
status_t relu_backward_device(tensor_t const d_dx,
                                  lcache_t* cache,
                                  tensor_t d_dy) {
  lcache_dump_stat(cache);
  tensor_t d_x = lcache_pop(cache);
  PINF("RELU_BACKWARD d_dx");
/*  print_tensor_device<<<1,1>>>(d_dy);*/
  print_tensor_device<<<1,1>>>(d_x);
  /*print_tensor_device<<<1,1>>>(d_dx);*/
  do_device_relu_backward<<<32, 1024>>>(d_dx, d_x, d_dy);
/*  print_tensor_device<<<1,1>>>(d_dy);*/
  /*print_tensor_device<<<1,1>>>(d_x);*/
  /*print_tensor_device<<<1,1>>>(d_dx);*/
  PINF("DEVICE");

  return S_OK;
}


status_t conv_relu_forward_device(hipblasHandle_t handle, tensor_t const d_x,
                                  tensor_t d_w, lcache_t* cache,
                                  conv_param_t const params, tensor_t d_y, struct layer_context_device * context) {
  AWNN_CHECK_EQ(d_x.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(d_w.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(d_y.mem_type, GPU_MEM);
  tensor_t d_tmp = context->d_tmp;

  print_tensor_device<<<1,1>>>(d_x);
  AWNN_CHECK_EQ(S_OK, convolution_forward_device(handle, d_x, d_w, cache, params, d_tmp));
  AWNN_CHECK_EQ(S_OK, relu_forward_device(d_tmp, cache, d_y));

  lcache_dump_stat(cache);
  return S_OK;
}

status_t conv_relu_backward_device(hipblasHandle_t handle, tensor_t d_dx,
                                   tensor_t d_dw, lcache_t* cache,
                                   conv_param_t const params,
                                   tensor_t const d_dy, struct layer_context_device * context) {
  AWNN_CHECK_EQ(d_dx.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(d_dw.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(d_dy.mem_type, GPU_MEM);
  status_t ret = S_ERR;

  PINF("CONV_RELU_BACKWARD");
  tensor_t d_tmp = context->d_dtmp;
  PINF("d_dy");
  lcache_dump_stat(cache);
  print_tensor_device<<<1,1>>>(d_dy);


  AWNN_CHECK_EQ(S_OK, relu_backward_device(d_tmp, cache, d_dy));
  PINF("AFTER relu backward: d_dtmp");
  print_tensor_device<<<1,1>>>(d_tmp);
  AWNN_CHECK_EQ(S_OK, convolution_backward_device(handle, d_dx, d_dw, cache, params, d_tmp));

  tensor_destroy_device(&d_tmp);
  ret = S_OK;
  return ret;
}

status_t conv_iden_relu_forward_device(
    hipblasHandle_t handle, tensor_t const d_x, tensor_t const d_iden,
    tensor_t d_w, lcache_t* cache, conv_param_t const params, tensor_t d_y) {
  AWNN_CHECK_EQ(d_x.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(d_iden.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(d_w.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(d_y.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(
      d_x.dim.dims[3],
      d_y.dim.dims[3]);  // in resnet tensor h/w doesn't change in each stage
  tensor_t d_tmp = tensor_make_alike_device(d_y);
  AWNN_CHECK_EQ(S_OK, convolution_forward_device(handle, d_x, d_w, cache, params, d_tmp));


  elementwise_add_inplace_device<<<32,1024>>>(d_tmp, d_iden);

  // tensor_elemwise_op_inplace(tmp, iden, TENSOR_OP_ADD);

  AWNN_CHECK_EQ(S_OK, relu_forward_device(d_tmp, cache, d_y));
  if(cache == NULL)
    tensor_destroy_device(&d_tmp);
  return S_OK;
}

status_t conv_iden_relu_backward_device(hipblasHandle_t handle, tensor_t d_dx,
                                        tensor_t d_diden, tensor_t d_dw,
                                        lcache_t* cache,
                                        conv_param_t const params,
                                        tensor_t const d_dy) {
  AWNN_CHECK_EQ(d_dx.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(d_diden.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(d_dw.mem_type, GPU_MEM);
  AWNN_CHECK_EQ(d_dy.mem_type, GPU_MEM);


  AWNN_CHECK_EQ(S_OK, relu_backward_device(d_diden, cache, d_dy));
  AWNN_CHECK_EQ(S_OK, convolution_backward_device(handle, d_dx, d_dw, cache, params, d_diden));

  return S_OK;
}

status_t resblock_forward_device(hipblasHandle_t handle, tensor_t const d_x,
                                 tensor_t d_w1, tensor_t d_w2, lcache_t* cache,
                                 conv_param_t const params, tensor_t d_y) {
  tensor_t d_tmp = tensor_make_alike_device(d_y);
  // TODO: pass context
  conv_relu_forward_device(handle, d_x, d_w1, cache, params, d_tmp, NULL);
  conv_iden_relu_forward_device(handle, d_tmp, d_x, d_w2, cache, params, d_y);

  tensor_destroy_device(&d_tmp);
  return S_OK;
}
status_t resblock_backward_device(hipblasHandle_t handle, tensor_t d_dx,
                                  tensor_t d_dw1, tensor_t d_dw2,
                                  lcache_t* cache, conv_param_t const params,
                                  tensor_t const d_dy) {
  tensor_t d_tmp = tensor_make_alike_device(d_dy);
  tensor_t d_dx_iden = tensor_make_alike_device(d_dx);

  conv_iden_relu_backward_device(handle, d_tmp, d_dx_iden, d_dw2, cache, params,
                                 d_dy);
  // TODO: pass context
  conv_relu_backward_device(handle, d_dx, d_dw1, cache, params, d_tmp, NULL);


  elementwise_add_inplace_device<<<32, 1024>>>(d_dx, d_dx_iden);
  // tensor_elemwise_op_inplace_device(d_dx, d_dx_iden, TENSOR_OP_ADD);
  tensor_destroy_device(&d_tmp);
  tensor_destroy_device(&d_dx_iden);

  return S_OK;
}
