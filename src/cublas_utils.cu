#include "awnndevice/cublas_utils.cuh"

static inline hipblasStatus_t cublasTgemm(hipblasHandle_t handle,
                           hipblasOperation_t op_a,
                           hipblasOperation_t op_b,
                           int m, int n, int k, const float * alpha,
                           const float *A, int lda, const float *B,
                           int ldb, const float * beta, float *C,
                           int ldc)
{
  return hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A,
                     lda, B, ldb, beta, C, ldc);
}


static inline hipblasStatus_t cublasTgemm(hipblasHandle_t handle,
                                  hipblasOperation_t op_a,
                                  hipblasOperation_t op_b,
                                  int m, int n, int k, const double * alpha,
                                  const double *A, int lda, const double *B,
                                  int ldb, const double * beta, double *C,
                                  int ldc)
{
  return hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A,
                     lda, B, ldb, beta, C, ldc);
}


//// transpose call double
//static inline hipblasStatus_t cublasTgeam(hipblasHandle_t handle,
//                                  hipblasOperation_t transa,
//                                  hipblasOperation_t transb,
//                                  int m, int n,
//                                  const double *alpha,
//                                  const double *A, int lda,
//                                  const double *beta,
//                                  const double *B, int ldb,
//                                  double *C, int ldc)
//{
//  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
//                     beta, B, ldb, C, ldc);
//}
//
//
//static hipblasStatus_t inline cublasTgeam(hipblasHandle_t handle,
//                                  hipblasOperation_t transa,
//                                  hipblasOperation_t transb,
//                                  int m, int n,
//                                  const float *alpha,
//                                  const float *A, int lda,
//                                  const float *beta,
//                                  const float *B, int ldb,
//                                  float *C, int ldc)
//{
//  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
//                     beta, B, ldb, C, ldc);
//}



/******************************************************************************
 * transpose (...)
 *
 * transposes a matrix and then returns its transpose as an rvalue reference
 *
 *  - this function uses the hipblasSgeam function to produce the transpose
 *    of M
 ******************************************************************************/
//tensor_t transpose_device (tensor_t src)
//{
//  assert(src.mem_type == GPU_MEM);
//
//  uint trans_shape[] = { src.dim.dims[1], src.dim.dims[0] };
//  tensor_t trans = tensor_make_device(trans_shape, ARRAY_SIZE(trans_shape));
//
//  T const alpha(1.0);
//  T const beta(0.0);
//
//  int m = src.dim.dims[0];
//  int n = src.dim.dims[1];
//
//  cublasTgeam(handle_, HIPBLAS_OP_T, HIPBLAS_OP_N,
//              m, n, &alpha,
//              src.data, n, &beta,
//              src.data, m,
//              trans.data, m
//  );
//
//  return trans;
//}

tensor_t cublas_gemm_launch(tensor_t A, tensor_t B) {
    const T alpha = 1.f;
    const T beta = 0.f;

    const int rowA = A.dim.dims[0];
    const int colA = A.dim.dims[1];
    const int colB = B.dim.dims[1];

    const T * srcA  = A.data;
    const T * srcB  = B.data;

    uint shape_res[] = { (uint)rowA, (uint)colB };
    tensor_t result = tensor_make_device(shape_res, ARRAY_SIZE(shape_res));
    T * out   = result.data;

    // Do the actual multiplication
    hipblasDgemm(handle_,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                colB,
                rowA,
                colA,
                &alpha,
                srcB,
                colB,
                srcA,
                colA,
                &beta,
                out,
                colB);
}