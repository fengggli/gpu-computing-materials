#include "hip/hip_runtime.h"
#include "awnn/layer_conv.h"
#include "cuda_defs.h"

/*
 * For the forward operation, there are a number of "hard" operations for the GPU.
 *
 * The primary one is the transformation of the input array to a 2D array through
 * the im2col process.  This process needs.
 *
 *    * padding
 *    * im2coll
 *
 * Additional critical operations that should be enabled on the GPU are
 *
 *    * tensor transpose 3012
 *    * dot product (gonna use cublas here most likely)
 */




/*
// used for forward
tensor_t tensor_make_transpose_3012(tensor_t t) {
  uint target_idx = 0;
  tensor_t tpose = tensor_make_copy(t);
  for (uint i = 0; i < t.dim.dims[3]; ++i) {  // for each of the new dim 0
    for (uint j = 0; j < t.dim.dims[0] * t.dim.dims[1] * t.dim.dims[2]; ++j) {
      tpose.data[target_idx++] = t.data[i + j * t.dim.dims[3]];
    }
  }
  uint const shape[] = { t.dim.dims[3], t.dim.dims[0], t.dim.dims[1], t.dim.dims[2] };
  tensor_reshape_(&tpose, shape, ARRAY_SIZE(shape));
  return tpose;
}

*/
static __global__ void _do_tensor_make_transpose_3012_device(tensor_t d_transpose, tensor_t d_src) {
  if (threadIdx.x == 0) {
    printf("entered _do_tensor_make_transpose_3012_device\n", threadIdx.x);
  }
}

tensor_t tensor_make_transpose_3012_device(tensor_t t) {
  uint const transposed_shape[] = { t.dim.dims[3], t.dim.dims[0], t.dim.dims[1], t.dim.dims[2] };

  tensor_t d_src = tensor_make_copy_h2d(t);

  tensor_t d_transposed = tensor_make_copy_h2d(t);
  tensor_reshape_(&d_transposed, transposed_shape, ARRAY_SIZE(transposed_shape));

  dim3 threads(32);
  dim3 blocks(1);
  PINF("device code is called");
  _do_tensor_make_transpose_3012_device<<<blocks, threads>>>(d_transposed, d_src);

  tensor_t h_transposed = tensor_make(transposed_shape, ARRAY_SIZE(transposed_shape));
  tensor_copy_d2h(h_transposed, d_transposed);
}



/**
 * TODO
 *
 * This function should just do the padding operation in parallel.  Although below in my notes of the
 * inner im2col, I speculate that this operation can be eliminated, I am still going to provide it in
 * the intitial cuda implementation
 */
static __global__ void _do_tensor_make_padded_square_input_device(tensor_t padded, tensor_t src, uint p, T val)
{
  if (threadIdx.x == 0) {
    printf("entered _do_tensor_make_padded_square_input_device\n", threadIdx.x);
  }
}

tensor_t tensor_make_padded_square_input_device(tensor_t t, uint p, T val) {

  uint padded_shape[] = { t.dim.dims[0], t.dim.dims[1], t.dim.dims[2] + 2 * p, t.dim.dims[3] + 2 * p };
  tensor_t d_padded = tensor_make_device(padded_shape, ARRAY_SIZE(padded_shape));
  tensor_t d_src = tensor_make_copy_h2d(t);

  dim3 threads(32);
  dim3 blocks(1);
  PINF("device code is called");

  _do_tensor_make_padded_square_input_device<<<blocks, threads>>>(d_padded, d_src, p, val);

  tensor_t h_padded = tensor_make(padded_shape, ARRAY_SIZE(padded_shape));
  tensor_copy_d2h(h_padded, d_padded);

  return h_padded;
}



static __global__ void _do_im2col_device(tensor_t const x, tensor_t const w, conv_param_t const params) {
  if (threadIdx.x == 0) {
    printf("entered _do_im2col_device\n", threadIdx.x);
  }
}

/*
 * This function just sets up the im2col.
 */
tensor_t im2col_device(tensor_t const x, tensor_t const w, conv_param_t const params)
{
  // TODO: make it handler lager size
  dim3 threads(32);
  dim3 blocks(1);
  PINF("device code is called");

  _do_im2col_device<<<blocks, threads>>>(x, w, params);
}




/**
 * TODO
 *
 * The inner should take the x_padded tensor and spread it out over the cols array
 * based on the stride and filter size.  Since we are doing the convolution with the
 * gemm approach, this is a core function.
 *
 * It should generally to how the convolution would work except that it is just doing
 * a copy instead of doing multiplication.  My suspicion is that the bulk of the
 * work from the convolution could be done here, and I'm not currently sure why
 * we would see an improvement in speed from GEMM.  Additionally, note that this
 * function takes in the padded array.  The padded array is used to allow a normal
 * stride in the loops in this fucntino, but it seems like the padded array could be
 * eliminated and this function could manage the padding elements as part of its
 * looping structure.
 *
 * If two loops are created (one for core elements, and one for padding elements)
 * a conditional could be avoided.
 */
static __global__ void _do_im2col_inner_device(tensor_t cols, tensor_t x_padded, uint N,  uint C,  uint H,  uint W,  uint HH, uint WW, uint filter_height, uint filter_width, uint padding, uint stride)
{
  printf("%d\n", threadIdx.x);
}

/**
 * im2col_inner_device is a setup function for the real call to actually launch the kernel.
 * For now, it will allocate and de-allocate / transfer mem to and from the GPU. In the pure
 * GPU based forward, this function will not be called, but rather the _do... function will be
 * called directly.
 */
status_t im2col_inner_device(tensor_t cols, tensor_t x_padded, uint N,  uint C,  uint H,  uint W,  uint HH, uint WW, uint filter_height, uint filter_width, uint padding, uint stride)
{

  tensor_t d_cols       = tensor_make_copy_h2d(cols);
  tensor_t d_x_padded   = tensor_make_copy_h2d(x_padded);

  // TODO: make it handler lager size
  dim3 threads(32);
  dim3 blocks(1);
  PINF("device code is called");

  _do_im2col_inner_device<<<blocks, threads>>>(d_cols, d_x_padded, N, C, H, W, HH, WW, filter_height, filter_width, padding, stride);

  tensor_copy_d2h(cols, d_cols);

  tensor_destroy_device(&d_cols);
  tensor_destroy_device(&d_x_padded);

  return S_ERR;
}




/*
 * Note that this is the only one that should likely remain global in the forward path.
 * The rest should become __device__ and should be called by this function
 */
static __global__ void _do_convolution_forward_device(tensor_t const x, tensor_t const w, lcache_t* cache, conv_param_t const params, tensor_t y)
{
  if (threadIdx.x == 0) {
    printf("entered _do_convolution_forward_device\n", threadIdx.x);
  }
}

/*
 * primary entry point for the forward function
 */
status_t convolution_forward_device(tensor_t const x, tensor_t const w, lcache_t* cache, conv_param_t const params, tensor_t y)
{

  dim3 threads(32);
  dim3 blocks(1);
  PINF("device code is called");

  _do_convolution_forward_device<<<blocks, threads>>>(x, w, cache, params, y);
  return S_ERR;
}








static __global__ void _do_convolution_backward_device(tensor_t dx, tensor_t dw, lcache_t* cache, conv_param_t const params, tensor_t const dout)
{
  if (threadIdx.x == 0) {
    printf("entered _do_col2im_inner_device\n");
  }
}

status_t convolution_backward_device(tensor_t dx, tensor_t dw, lcache_t* cache, conv_param_t const params, tensor_t const dout)
{
  dim3 threads(32);
  dim3 blocks(1);
  PINF("device code is called");
  _do_convolution_backward_device<<<blocks, threads>>>(dx, dw, cache, params, dout);
}



static __global__ void _do_col2im_device(tensor_t cols, uint N, uint C, uint H, uint W, uint field_height, uint field_width, uint padding, uint stride)
{
  if (threadIdx.x == 0) {
    printf("entered _do_col2im_inner_device\n");
  }
}

tensor_t col2im_device(tensor_t cols, uint N, uint C, uint H, uint W, uint field_height, uint field_width, uint padding, uint stride)
{
  dim3 threads(32);
  dim3 blocks(1);
  PINF("device code is called");
  _do_col2im_device<<<blocks, threads>>>(cols, N, C, H, W, field_height, field_width, padding, stride);
}



static __global__ void _do_col2im_inner_device(tensor_t cols, tensor_t x_padded, uint N, uint C, uint H, uint W, uint HH, uint WW, uint field_height, uint field_width, uint padding, uint stride)
{
  if (threadIdx.x == 0) {
    printf("entered _do_col2im_inner_device\n");
  }
}

void col2im_inner_device(tensor_t cols, tensor_t x_padded, uint N, uint C, uint H, uint W, uint HH, uint WW, uint field_height, uint field_width, uint padding, uint stride) {
  // TODO: make it handler lager size
  dim3 threads(32);
  dim3 blocks(1);
  PINF("device code is called");
  _do_col2im_inner_device<<<blocks, threads>>>(cols, x_padded, N, C, H, W, HH, WW, field_height, field_width, padding, stride);
}
