#include "hip/hip_runtime.h"
#include "awnn/channel.h"
#include "awnn/layer_pool.h"

/* this is a */
static __global__ void _do_forward(T *input, uint num_image, uint num_channels,
                                   uint channel_capacity, T *output) {
  // do nothing
  /*  for (uint i = 0; i < num_images; ++i)*/
  /*for (uint j = 0; j < num_channels; ++j) {*/
  /*double mean = 0;*/
  /*for()*/
  /*y.data[i * num_channels + j] =*/
  /*scan(x.data + i * num_channels * channel_capacity +*/
  /*j * channel_capacity,*/
  /*channel_capacity(x));*/
  /*}*/
}

// y: N, C, 1, 1
status_t global_avg_pool_forward_device(tensor_t const x, lcache_t *cache,
                                        tensor_t y) {
  uint num_images = x.dim.dims[0];
  uint num_channels = x.dim.dims[1];
  T *d_x, *d_y;

  hipMalloc(&d_x, sizeof(T) * tensor_get_capacity(x));
  hipMalloc(&d_y, sizeof(T) * tensor_get_capacity(y));
  // _do_forward(d_x, num_image, num_channels, channel_capacity(x), dout);

  // tensor_t x_d = tensor_make_copy_h2d();

  // create cache
  if (cache) {
    tensor_t t = tensor_make_empty_with_dim(x.dim);
    lcache_push(cache, t);
  }
  hipFree(d_x);
  hipFree(d_y);

  return S_OK;
}

status_t global_avg_pool_backward_device(tensor_t dx, lcache_t *cache,
                                         tensor_t const dy) {
  tensor_t t = lcache_pop(cache);
  uint N = t.dim.dims[0];
  uint C = t.dim.dims[1];
  uint H = t.dim.dims[2];
  uint W = t.dim.dims[3];

  float scale_by = 1.0 / (H * W);
  //  tensor_t scales = tensor_make_scalar_alike(t, scale_by);

  //  tensor_elemwise_op_inplace(scales, dy, TENSOR_OP_MUL);
  for (uint i = 0; i < N; ++i)
    for (uint j = 0; j < C; ++j)
      for (uint k = 0; k < H; ++k)
        for (uint l = 0; l < W; ++l)
          dx.data[i * C * H * W + j * H * W + k * W + l] =
              scale_by * dy.data[i * C + j];

  // free layer cache
  tensor_destroy(t);

  return S_OK;
}
