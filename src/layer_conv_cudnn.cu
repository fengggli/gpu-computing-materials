/*
 * Description:
 *
 * Author: Yuankun Fu
 * e-mail: qoofyk@gmail.com
 */

#include "awnn/tensor.h"
#include "awnn/layer.h"
#include "awnn/layer_conv.h"

#include "awnn/layer_cudnn.h"
#define THRESHOLD               2.0e-2

//#define PRINT_VERBOSE

static void generateStrides(const int* dimA, int* strideA, int nbDims, hipdnnTensorFormat_t filterFormat) {
  //For INT8x4 and INT8x32 we still compute standard strides here to input
  //into the cuDNN functions. We will manually scale by resizeFactor in the cpu ref.
  if (filterFormat == HIPDNN_TENSOR_NCHW || filterFormat == HIPDNN_TENSOR_NCHW_VECT_C) {
    strideA[nbDims-1] = 1 ;
    for(int d = nbDims-2 ; d >= 0 ; d--) {
      strideA[d] = strideA[d+1] * dimA[d+1] ;
    }
  } else {
    //Here we assume that the format is HIPDNN_TENSOR_NHWC
    strideA[1] = 1;
    strideA[nbDims-1] = strideA[1]*dimA[1];
    for(int d = nbDims-2 ; d >= 2 ; d--) {
      strideA[d] = strideA[d+1] * dimA[d+1] ;
    }
    strideA[0] = strideA[2]*dimA[2];
  }
}

template <typename T_ELEM>
status_t doForward(tensor_t const x, tensor_t const w, tensor_t y, int* dimA,
                   int* padA, int* convstrideA, int* filterdimA, hipdnnTensorFormat_t filterFormat, hipdnnDataType_t dataType,
                   int mathType,
                   hipdnnHandle_t handle_, hipdnnTensorDescriptor_t cudnnIdesc,
                   hipdnnFilterDescriptor_t cudnnFdesc,
                   hipdnnTensorDescriptor_t cudnnOdesc,
                   hipdnnConvolutionDescriptor_t cudnnConvDesc) {
  T_ELEM* devPtrI=x.data;
  T_ELEM* devPtrF=w.data;
  T_ELEM* devPtrO=y.data;

  void *workSpace = 0;
  size_t workSpaceSize;

  hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
//  hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_FFT;
  /*hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING;*/
  /*hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;*/

  int convDim = 2;

  float alpha = 1.0f;
  float beta = 0.0;
  int dilationA[] = {1, 1};

  int* dimA_padded = dimA;
  int* outdimA_padded = (int*) (y.dim.dims);
  int* filterdimA_padded = filterdimA;
  int strideA_padded[4];
  int outstrideA_padded[4];
  int filterstrideA_padded[4];

#ifdef PRINT_VERBOSE
  PDBG("====USER DIMENSIONS====\n");
  PDBG("input dims are %d, %d, %d, %d\n", dimA[0], dimA[1], dimA[2], dimA[3]);
  PDBG("filter dims are %d, %d, %d, %d\n", filterdimA[0], filterdimA[1],
       filterdimA[2], filterdimA[3]);
  PDBG("output dims are %d, %d, %d, %d\n", outdimA[0], outdimA[1], outdimA[2],
       outdimA[3]);
  PDBG("====PADDING DIMENSIONS====\n");
  PDBG("padded input dims are %d, %d, %d, %d\n", dimA_padded[0], dimA_padded[1],
       dimA_padded[2], dimA_padded[3]);
  PDBG("padded filter dims are %d, %d, %d, %d\n", filterdimA_padded[0],
       filterdimA_padded[1], filterdimA_padded[2], filterdimA_padded[3]);
  PDBG("padded output dims are %d, %d, %d, %d\n", outdimA_padded[0],
       outdimA_padded[1], outdimA_padded[2], outdimA_padded[3]);
#endif

  generateStrides(dimA_padded, strideA_padded, 4, filterFormat);
  generateStrides(filterdimA_padded, filterstrideA_padded, 4, filterFormat);
  generateStrides(outdimA_padded, outstrideA_padded, 4, filterFormat);

  checkCudnnErr( hipdnnSetTensorNdDescriptor(cudnnIdesc, dataType, convDim+2, dimA_padded, strideA_padded) );
  checkCudnnErr( hipdnnSetTensorNdDescriptor(cudnnOdesc, dataType, convDim+2, outdimA_padded, outstrideA_padded) );
  checkCudnnErr( hipdnnSetConvolutionNdDescriptor(cudnnConvDesc,
                                                 convDim,
                                                 padA,
                                                 convstrideA,
                                                 dilationA,
                                                 HIPDNN_CONVOLUTION, dataType));

  checkCudnnErr( hipdnnSetFilterNdDescriptor(cudnnFdesc, dataType, filterFormat, convDim+2, filterdimA_padded));

  if (mathType == 1) {
    checkCudnnErr( hipdnnSetConvolutionMathType(cudnnConvDesc, HIPDNN_TENSOR_OP_MATH) );
  }

  // start computation of cudnn forward
  checkCudnnErr ( hipdnnGetConvolutionForwardWorkspaceSize(handle_, cudnnIdesc, cudnnFdesc, cudnnConvDesc,
                                                          cudnnOdesc, algo, &workSpaceSize) );

  if (workSpaceSize > 0) {
    hipMalloc(&workSpace, workSpaceSize);
  }

  checkCudnnErr ( hipdnnConvolutionForward (handle_,
                                           (void*)(&alpha),
                                           cudnnIdesc, devPtrI,
                                           cudnnFdesc, devPtrF,
                                           cudnnConvDesc,
                                           algo,
                                           workSpace, workSpaceSize,
                                           (void*)(&beta),
                                           cudnnOdesc, devPtrO) );
  checkCudaErr( hipDeviceSynchronize() );

clean:
  if (workSpace) hipFree(workSpace);

  return S_OK;
}

template <typename T_ELEM>
status_t doBackward(tensor_t x, tensor_t dx, tensor_t w, tensor_t dw,
                    tensor_t const dout, int* dimA, int* padA, int* convstrideA, int* filterdimA,
                    hipdnnTensorFormat_t filterFormat,
                    hipdnnDataType_t dataType, int mathType,
                    hipdnnHandle_t handle_, hipdnnTensorDescriptor_t cudnnIdesc,
                    hipdnnFilterDescriptor_t cudnnFdesc,
                    hipdnnTensorDescriptor_t cudnnOdesc,
                    hipdnnConvolutionDescriptor_t cudnnConvDesc) {
  T_ELEM* devPtr_dx = dx.data;
  T_ELEM* devPtr_w = w.data;

  T_ELEM* devPtr_x = x.data;
  T_ELEM* devPtr_dw = dw.data;

  T_ELEM* devPtrO = dout.data;

  hipdnnConvolutionBwdDataAlgo_t algo_data = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1;
  hipdnnConvolutionBwdFilterAlgo_t algo_weight = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1;

  void *workSpace = 0;
  size_t workSpaceSize;

  int convDim = 2;

  float alpha = 1.0f;
  float beta = 0.0;
  int dilationA[] = {1, 1};
//  int outdimA[4];

  int* dimA_padded = dimA;
  int* outdimA_padded = (int*) (dout.dim.dims);
  int* filterdimA_padded = filterdimA;

  int strideA_padded[4];
  int outstrideA_padded[4];
  int filterstrideA_padded[4];

#ifdef PRINT_VERBOSE
  PDBG("====USER DIMENSIONS====\n");
  PDBG("input dims are %d, %d, %d, %d\n", dimA[0], dimA[1], dimA[2], dimA[3]);
  PDBG("filter dims are %d, %d, %d, %d\n", filterdimA[0], filterdimA[1],
       filterdimA[2], filterdimA[3]);
  PDBG("output dims are %d, %d, %d, %d\n", outdimA[0], outdimA[1], outdimA[2],
       outdimA[3]);
  PDBG("====PADDING DIMENSIONS====\n");
  PDBG("padded input dims are %d, %d, %d, %d\n", dimA_padded[0], dimA_padded[1],
       dimA_padded[2], dimA_padded[3]);
  PDBG("padded filter dims are %d, %d, %d, %d\n", filterdimA_padded[0],
       filterdimA_padded[1], filterdimA_padded[2], filterdimA_padded[3]);
  PDBG("padded output dims are %d, %d, %d, %d\n", outdimA_padded[0],
       outdimA_padded[1], outdimA_padded[2], outdimA_padded[3]);
#endif

  generateStrides(dimA_padded, strideA_padded, 4, filterFormat);
  generateStrides(filterdimA_padded, filterstrideA_padded, 4, filterFormat);
  generateStrides(outdimA_padded, outstrideA_padded, 4, filterFormat);

  checkCudnnErr( hipdnnSetTensorNdDescriptor(cudnnIdesc, dataType, convDim+2, dimA_padded, strideA_padded) );
  checkCudnnErr( hipdnnSetTensorNdDescriptor(cudnnOdesc, dataType, convDim+2, outdimA_padded, outstrideA_padded) );
  checkCudnnErr( hipdnnSetConvolutionNdDescriptor(cudnnConvDesc,
                                                 convDim,
                                                 padA,
                                                 convstrideA,
                                                 dilationA,
                                                 HIPDNN_CONVOLUTION, dataType));

  checkCudnnErr( hipdnnSetFilterNdDescriptor(cudnnFdesc, dataType, filterFormat, convDim+2, filterdimA_padded));

  if (mathType == 1) {
    checkCudnnErr( hipdnnSetConvolutionMathType(cudnnConvDesc, HIPDNN_TENSOR_OP_MATH) );
  }

  // start compute cudnn backward data
  checkCudnnErr ( hipdnnGetConvolutionBackwardDataWorkspaceSize(handle_, cudnnFdesc, cudnnOdesc, cudnnConvDesc,
                                                               cudnnIdesc, algo_data, &workSpaceSize) );

  if (workSpaceSize > 0) {
    hipMalloc(&workSpace, workSpaceSize);
  }

  checkCudnnErr ( hipdnnConvolutionBackwardData (handle_,
                                                (void*)(&alpha),
                                                cudnnFdesc, devPtr_w,
                                                cudnnOdesc, devPtrO,
                                                cudnnConvDesc,
                                                algo_data,
                                                workSpace, workSpaceSize,
                                                (void*)(&beta),
                                                cudnnIdesc, devPtr_dx) );
  checkCudaErr( hipDeviceSynchronize() );

  // free workSpace
  if (workSpace) hipFree(workSpace);

  // start compute cudnn backward filter
  checkCudnnErr ( hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle_, cudnnIdesc, cudnnOdesc, cudnnConvDesc,
                                                                 cudnnFdesc, algo_weight, &workSpaceSize) );

  if (workSpaceSize > 0) {
    hipMalloc(&workSpace, workSpaceSize);
  }

  checkCudnnErr ( hipdnnConvolutionBackwardFilter (handle_,
                                                  (void*)(&alpha),
                                                  cudnnIdesc, devPtr_x,
                                                  cudnnOdesc, devPtrO,
                                                  cudnnConvDesc,
                                                  algo_weight,
                                                  workSpace, workSpaceSize,
                                                  (void*)(&beta),
                                                  cudnnFdesc, devPtr_dw) );
  checkCudaErr( hipDeviceSynchronize() );


clean:
  if (workSpace) hipFree(workSpace);
  return S_OK;
}


status_t convolution_forward_cudnn(tensor_t const x, tensor_t const w, lcache_t* cache,
                                   conv_param_t const params, tensor_t y,
                                   hipdnnHandle_t handle_, hipdnnTensorDescriptor_t cudnnIdesc,
                                   hipdnnFilterDescriptor_t cudnnFdesc,
                                   hipdnnTensorDescriptor_t cudnnOdesc,
                                   hipdnnConvolutionDescriptor_t cudnnConvDesc){
  int mathType = 0;  // 0: HIPDNN_DEFAULT_MATH -> Tensor Core Operations are not
                     // used 1: HIPDNN_TENSOR_OP_MATH -> The use of Tensor Core
                     // Operations is permitted.

  int* dimA = (int*)(x.dim.dims);  // N, C, H, W;
  int padA[] = {(int)params.padding, (int)params.padding};
  int convstrideA[] = {(int)params.stride, (int)params.stride};
  // batch size and feature layers must be multiples of 4 or 32 when using int8x4 or int8x32 respectively
  int* filterdimA = (int*)(w.dim.dims);  // k, c, r, s //k, c, r, s

  hipdnnTensorFormat_t  filterFormat = HIPDNN_TENSOR_NCHW;

#ifdef PRINT_VERBOSE
  PDBG("Testing using cudnn forward\n");
#endif

#ifdef AWNN_USE_FLT32
  status_t ret =
      doForward<T>(x, w, y, dimA, padA, convstrideA, filterdimA, filterFormat,
                   HIPDNN_DATA_FLOAT, mathType,
                   handle_, cudnnIdesc, cudnnFdesc, cudnnOdesc, cudnnConvDesc);
#else
  status_t ret =
      doForward<T>(x, w, y, dimA, padA, convstrideA, filterdimA, filterFormat,
                   HIPDNN_DATA_DOUBLE, mathType,
                   handle_, cudnnIdesc, cudnnFdesc, cudnnOdesc, cudnnConvDesc);
#endif

  // shadow copy
  tensor_t cached_x_shadow = x;
  tensor_t cached_w_shadow = w;

  // TODO put w and data
  if (cache) {
    lcache_push(cache, cached_x_shadow);
    lcache_push(cache, cached_w_shadow);
  }
  return ret;
}

status_t convolution_backward_cudnn(tensor_t dx, tensor_t dw, lcache_t* cache,
                                    conv_param_t const params, tensor_t const dout,
                                    hipdnnHandle_t handle_, hipdnnTensorDescriptor_t cudnnIdesc,
                                    hipdnnFilterDescriptor_t cudnnFdesc,
                                    hipdnnTensorDescriptor_t cudnnOdesc,
                                    hipdnnConvolutionDescriptor_t cudnnConvDesc) {
  tensor_t x, w;

  // NOTE : the order of pop matters, should be flattened_x, w, x (reverse of
  // forward)
  w = lcache_pop(cache);
  x = lcache_pop(cache);

  int mathType = 0;

  int* dimA = (int*)(dx.dim.dims);  // N, C, H, W;
  int padA[] = {(int)params.padding, (int)params.padding};
  int convstrideA[] = {(int)params.stride, (int)params.stride};
  //batch size and feature layers must be multiples of 4 or 32 when using int8x4 or int8x32 respectively
  int* filterdimA = (int*)(w.dim.dims);  // k, c, r, s

  hipdnnTensorFormat_t  filterFormat = HIPDNN_TENSOR_NCHW;

#ifdef PRINT_VERBOSE
  PDBG("Testing using cudnn backward data\n");
#endif

#ifdef AWNN_USE_FLT32
  status_t ret =
      doBackward<T>(x, dx, w, dw, dout, dimA, padA, convstrideA, filterdimA, filterFormat,
          HIPDNN_DATA_FLOAT, mathType,
          handle_, cudnnIdesc, cudnnFdesc, cudnnOdesc, cudnnConvDesc);
#else
  status_t ret =
      doBackward<T>(x, dx, w, dw, dout, dimA, padA, convstrideA, filterdimA, filterFormat,
                    HIPDNN_DATA_DOUBLE, mathType,
                    handle_, cudnnIdesc, cudnnFdesc, cudnnOdesc, cudnnConvDesc);
#endif

  return ret;
}
